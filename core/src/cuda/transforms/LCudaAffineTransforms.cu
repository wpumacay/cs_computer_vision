#include "hip/hip_runtime.h"


#include "../../../include/cuda/transforms/LCudaAffineTransforms.h"

using namespace std;

__device__ float device_interpolate_nn( float* d_mat,
                                        int cols, int rows, int channels,
                                        float fx_c, float fy_c, int pChannel )
{

    // 1. fx, fy are centric coordinates, transform the back to the original system
    float _fx = fx_c + 0.5 * cols;
    float _fy = 0.5 * rows - fy_c;

    // 2. Apply nn interpolation
    int _x = __float2int_ru( _fx );
    int _y = __float2int_ru( _fy );

    if ( _x < 0 || _x > cols - 1 ||
         _y < 0 || _y > rows - 1 )
    {
        return 0.0f;
    }

    return d_mat[ channels * ( _x + _y * cols ) + pChannel ];
}



__global__ void kernel_warpAffine( float* d_srcMat,
                                   float* d_dstMat,
                                   int srcCols, int srcRows,
                                   int dstCols, int dstRows,
                                   int channels,
                                   float* d_matrix3f )
{

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ( x > dstCols - 1 ||
         y > dstRows - 1 )
    {
        return;
    }

    // 1. Transform it to image centric coordinates
    float _xc = x - 0.5 * dstCols;
    float _yc = 0.5 * dstRows - y;

    // 2. Get source coordinates using the inverse transform
    float _x = d_matrix3f[0] * _xc + d_matrix3f[1] * _yc + d_matrix3f[2];
    float _y = d_matrix3f[3] * _xc + d_matrix3f[4] * _yc + d_matrix3f[5];

    // 3. Apply interpolation of the pixel at _x, _y ...
    // in the src image for each channel
    for ( int c = 0; c < channels; c++ )
    {
        float _v = device_interpolate_nn( d_srcMat, 
                                          srcCols, srcRows, channels,
                                          _x, _y, c );

        d_dstMat[ channels * ( x + y * dstCols ) + c ] = _v;
    }

}









namespace felix
{


    namespace cuda
    {


        core::LCudaMat LCudaAffineTransforms::warpAffine( core::LCudaMat& lmat,
                                                          int pDstCols, int pDstRows,
                                                          float* pMatrix3f )
        {

            // Calculate split size

            int _bx = ceil( ( float ) pDstCols / CUDA_THREADS_SPLIT_2D );
            int _by = ceil( ( float ) pDstRows / CUDA_THREADS_SPLIT_2D );

            //cout << "partition: " << _bx << " - " << _by << endl;

            core::LCudaMat _res( pDstRows, pDstCols, lmat.channels() );

            lmat.loadToGpu();
            _res.loadToGpu();

            const dim3 _blockSize( CUDA_THREADS_SPLIT_2D, CUDA_THREADS_SPLIT_2D );
            const dim3 _gridSize( _bx, _by );

            // Inverse by cofactors

            float _det = pMatrix3f[0] * pMatrix3f[4] - pMatrix3f[3] * pMatrix3f[1];

            float _c11 = pMatrix3f[4] / _det;
            float _c12 = -pMatrix3f[3] / _det;
            float _c13 = 0.0f;

            float _c21 = -pMatrix3f[1] / _det;
            float _c22 = pMatrix3f[0] / _det;
            float _c23 = 0.0f;            

            float _c31 = ( pMatrix3f[1] * pMatrix3f[5] - pMatrix3f[2] * pMatrix3f[4] ) / _det;
            float _c32 = -( pMatrix3f[0] * pMatrix3f[5] - pMatrix3f[2] * pMatrix3f[3] ) / _det;
            float _c33 = 1.0f;

            //cout << "_det: " << _det << endl;

            float _pMatInv[9] = { _c11, _c21, _c31,
                                  _c12, _c22, _c32,
                                  _c13, _c23, _c33 };

            // Send inverse to gpu
            float* d_matrix3f;
            hipMalloc( ( void** ) &d_matrix3f, sizeof( float ) * 9 );
            hipMemcpy( d_matrix3f, _pMatInv, sizeof( float ) * 9, hipMemcpyHostToDevice );

            kernel_warpAffine<<< _gridSize, _blockSize >>>( lmat.devGpuBuffer(),
                                                            _res.devGpuBuffer(),
                                                            lmat.cols(), lmat.rows(), 
                                                            _res.cols(), _res.rows(),
                                                            lmat.channels(),
                                                            d_matrix3f );

            _res.loadFromGpu();

            hipFree( d_matrix3f );

            return _res;
        }

        core::LCudaMat LCudaAffineTransforms::scale( core::LCudaMat& lCudaMat,
                                                     int pDstCols, int pDstRows,
                                                     float sx, float sy )
        {
            float* _matrix3f = new float[9];

            _matrix3f[0] = sx;   _matrix3f[1] = 0.0f; _matrix3f[2] = 0.0f;
            _matrix3f[3] = 0.0f; _matrix3f[4] = sy;   _matrix3f[5] = 0.0f;
            _matrix3f[6] = 0.0f; _matrix3f[7] = 0.0f; _matrix3f[8] = 1.0f;

            core::LCudaMat _res = LCudaAffineTransforms::warpAffine( lCudaMat, pDstCols, pDstRows, _matrix3f );

            delete[] _matrix3f;

            return _res;
        }

        core::LCudaMat LCudaAffineTransforms::rotate( core::LCudaMat& lCudaMat,
                                                     int pDstCols, int pDstRows,
                                                     float angle )
        {
            float* _matrix3f = new float[9];

            float _c = cos( angle );
            float _s = sin( angle );

            _matrix3f[0] = _c;   _matrix3f[1] = -_s;  _matrix3f[2] = 0.0f;
            _matrix3f[3] = _s;   _matrix3f[4] = _c;   _matrix3f[5] = 0.0f;
            _matrix3f[6] = 0.0f; _matrix3f[7] = 0.0f; _matrix3f[8] = 1.0f;

            core::LCudaMat _res = LCudaAffineTransforms::warpAffine( lCudaMat, pDstCols, pDstRows, _matrix3f );

            delete[] _matrix3f;

            return _res;
        }

        core::LCudaMat LCudaAffineTransforms::translate( core::LCudaMat& lCudaMat,
                                                     int pDstCols, int pDstRows,
                                                     float tx, float ty )
        {
            float* _matrix3f = new float[9];

            _matrix3f[0] = 1.0f; _matrix3f[1] = 0.0f; _matrix3f[2] = tx;
            _matrix3f[3] = 0.0f; _matrix3f[4] = 1.0f; _matrix3f[5] = ty;
            _matrix3f[6] = 0.0f; _matrix3f[7] = 0.0f; _matrix3f[8] = 1.0f;

            core::LCudaMat _res = LCudaAffineTransforms::warpAffine( lCudaMat, pDstCols, pDstRows, _matrix3f );

            delete[] _matrix3f;

            return _res;
        }

    }





}