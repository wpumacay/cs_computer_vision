#include "hip/hip_runtime.h"


#include "../../../include/cuda/testing/LCudaTesting.h"

using namespace std;


__global__ void kernel_test_rgb( float* d_Mat,
                                 int cols, int rows, 
                                 int channels )
{

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ( x > cols - 1 ||
         y > rows - 1 )
    {
        return;
    }

    for ( int c = 0; c < channels; c++ )
    {
        if ( c == 0 )
        {
            d_Mat[ channels * ( x + y * cols ) + c ] = 1.0f;
        }
        else
        {
            d_Mat[ channels * ( x + y * cols ) + c ] = 0.0f;
        }
            
    }
}





namespace felix
{


    namespace cuda
    {


        core::LCudaMat LCudaTesting::test_gen_rgb( int rows, int cols, int channels )
        {

            // Calculate split size

            int _bx = ceil( ( float ) cols / CUDA_THREADS_SPLIT_2D );
            int _by = ceil( ( float ) rows / CUDA_THREADS_SPLIT_2D );

            core::LCudaMat _res( rows, cols, channels );
            _res.loadToGpu();

            const dim3 _blockSize( CUDA_THREADS_SPLIT_2D, CUDA_THREADS_SPLIT_2D );
            const dim3 _gridSize( _bx, _by );

            kernel_test_rgb<<< _gridSize, _blockSize >>>( _res.devGpuBuffer(),
                                                          _res.cols(), _res.rows(), 
                                                          _res.channels() );

            _res.loadFromGpu();

            return _res;
        }



    }





}